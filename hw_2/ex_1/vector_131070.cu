#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>


#define N 1024
#define MAX_ERR 1e-6

__global__ void vector_add(float *out, float *a, float *b, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) {
        out[tid] = a[tid] + b[tid];
    }
}

int main(){
    float *a, *b, *out;
    float *d_a, *d_b, *d_out; 
    hipEvent_t start, stop;
    float time;

    // Allocate host memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize host arrays
    for(int i = 0; i < N; i++){
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    // Allocate device memory 
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    // Create events
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Start recording the time for host to device data transfer
    hipEventRecord(start, 0);

    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    // Stop recording the time and calculate the elapsed time
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Time for host to device data transfer: %f ms\n", time);

    // Executing kernel 
    int block_size = 256;
    int grid_size = ((N + block_size - 1) / block_size);

    // Start recording the time for kernel execution
    hipEventRecord(start, 0);

    vector_add<<<grid_size,block_size>>>(d_out, d_a, d_b, N);

    // Stop recording the time and calculate the elapsed time
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Time for CUDA kernel execution: %f ms\n", time);

    // Start recording the time for device to host data transfer
    hipEventRecord(start, 0);

    // Transfer data back to host memory
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

    // Stop recording the time and calculate the elapsed time
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Time for device to host data transfer: %f ms\n", time);

    // Verification
    for(int i = 0; i < N; i++){
        assert(fabs(out[i] - a[i] - b[i]) < MAX_ERR);
    }

    printf("PASSED\n");

    // Deallocate device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    // Deallocate host memory
    free(a); 
    free(b); 
    free(out);

    // Destroy events
    hipEventDestroy(start);
    hipEventDestroy(stop);
}


